#include "hip/hip_runtime.h"
#include "gelu_cuda.h"
#include <cmath>
#include <hip/hip_runtime.h>

// Ядро CUDA для вычисления GELU
__global__ void GeluKernel(const float* input, float* output, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        output[idx] = 0.5f * x * (1.0f + tanh(sqrt(2.0f / M_PI) * (x + 0.044715f * x * x * x)));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    size_t v_size = input.size();
    std::vector<float> output(v_size);

    float* d_input;
    float* d_output;
    hipMalloc(&d_input, v_size * sizeof(float));
    hipMalloc(&d_output, v_size * sizeof(float));

    hipMemcpy(d_input, input.data(), v_size * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (v_size + blockSize - 1) / blockSize;

    GeluKernel<<<numBlocks, blockSize>>>(d_input, d_output, v_size);

    hipMemcpy(output.data(), d_output, v_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}
